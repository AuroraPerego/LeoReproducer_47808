#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdint>
#include <iostream>
#include <random>
#include <vector>

#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/requireDevices.h"
#include "HeterogeneousCore/CUDAUtilities/interface/launch.h"
#include "HeterogeneousCore/CUDAUtilities/interface/currentDevice.h"
// PixelTrackUtilities only included in order to compile SoALayout with Eigen columns
#include "CUDADataFormats/Track/interface/PixelTrackUtilities.h"
#include "CUDADataFormats/Vertex/interface/ZVertexUtilities.h"
#include "CUDADataFormats/Vertex/interface/ZVertexSoAHeterogeneousHost.h"
#include "CUDADataFormats/Vertex/interface/ZVertexSoAHeterogeneousDevice.h"

#include "RecoVertex/PixelVertexFinding/plugins/PixelVertexWorkSpaceUtilities.h"
#include "RecoVertex/PixelVertexFinding/plugins/PixelVertexWorkSpaceSoAHost.h"
#include "RecoVertex/PixelVertexFinding/plugins/PixelVertexWorkSpaceSoADevice.h"
#include "RecoVertex/PixelVertexFinding/plugins/gpuClusterTracksByDensity.h"
#include "RecoVertex/PixelVertexFinding/plugins/gpuFitVertices.h"
#include "RecoVertex/PixelVertexFinding/plugins/gpuSortByPt2.h"
#include "RecoVertex/PixelVertexFinding/plugins/gpuSplitVertices.h"

struct Event {
  std::vector<float> zvert;
  std::vector<uint16_t> itrack;
  std::vector<float> ztrack;
  std::vector<float> eztrack;
  std::vector<float> pttrack;
  std::vector<uint16_t> ivert;
};

struct ClusterGenerator {
  explicit ClusterGenerator(float nvert, float ntrack)
      : rgen(-13., 13), errgen(0.005, 0.025), clusGen(nvert), trackGen(ntrack), gauss(0., 1.), ptGen(1.) {}

  void operator()(Event& ev) {
    int nclus = clusGen(reng);
    ev.zvert.resize(nclus);
    ev.itrack.resize(nclus);
    for (auto& z : ev.zvert) {
      z = 3.5f * gauss(reng);
    }

    ev.ztrack.clear();
    ev.eztrack.clear();
    ev.ivert.clear();
    ev.pttrack.clear();
    for (int iv = 0; iv < nclus; ++iv) {
      auto nt = trackGen(reng);
      ev.itrack[iv] = nt;
      for (int it = 0; it < nt; ++it) {
        auto err = errgen(reng);  // reality is not flat....
        ev.ztrack.push_back(ev.zvert[iv] + err * gauss(reng));
        ev.eztrack.push_back(err * err);
        ev.ivert.push_back(iv);
        ev.pttrack.push_back((iv == 5 ? 1.f : 0.5f) + ptGen(reng));
        ev.pttrack.back() *= ev.pttrack.back();
      }
    }
    // add noise
    auto nt = 2 * trackGen(reng);
    for (int it = 0; it < nt; ++it) {
      auto err = 0.03f;
      ev.ztrack.push_back(rgen(reng));
      ev.eztrack.push_back(err * err);
      ev.ivert.push_back(9999);
      ev.pttrack.push_back(0.5f + ptGen(reng));
      ev.pttrack.back() *= ev.pttrack.back();
    }
  }

  std::mt19937 reng;
  std::uniform_real_distribution<float> rgen;
  std::uniform_real_distribution<float> errgen;
  std::poisson_distribution<int> clusGen;
  std::poisson_distribution<int> trackGen;
  std::normal_distribution<float> gauss;
  std::exponential_distribution<float> ptGen;
};

__global__ void print(gpuVertexFinder::VtxSoAView pdata, gpuVertexFinder::WsSoAView pws) {
  auto& __restrict__ ws = pws;
  printf("nt,nv %d %d,%d\n", ws.ntrks(), pdata.nvFinal(), ws.nvIntermediate());
}

int main() {
  hipStream_t stream;
  cms::cudatest::requireDevices();
  cudaCheck(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  ZVertexSoADevice onGPU_d(stream);
  gpuVertexFinder::workSpace::PixelVertexWorkSpaceSoADevice ws_d(stream);

  Event ev;

  float eps = 0.1f;
  std::array<float, 3> par{{eps, 0.01f, 9.0f}};
  for (int nav = 30; nav < 80; nav += 20) {
    ClusterGenerator gen(nav, 10);

    for (int i = 8; i < 20; ++i) {
      auto kk = i / 4;  // M param

      gen(ev);

      gpuVertexFinder::init<<<1, 1, 0, stream>>>(onGPU_d.view(), ws_d.view());

      std::cout << "v,t size " << ev.zvert.size() << ' ' << ev.ztrack.size() << std::endl;
      auto nt = ev.ztrack.size();
      cudaCheck(hipMemcpy(&ws_d.view().ntrks(), &nt, sizeof(uint32_t), hipMemcpyHostToDevice));
      cudaCheck(
          hipMemcpy(ws_d.view().zt(), ev.ztrack.data(), sizeof(float) * ev.ztrack.size(), hipMemcpyHostToDevice));
      cudaCheck(
          hipMemcpy(ws_d.view().ezt2(), ev.eztrack.data(), sizeof(float) * ev.eztrack.size(), hipMemcpyHostToDevice));
      cudaCheck(
          hipMemcpy(ws_d.view().ptt2(), ev.pttrack.data(), sizeof(float) * ev.eztrack.size(), hipMemcpyHostToDevice));

      std::cout << "M eps, pset " << kk << ' ' << eps << ' ' << (i % 4) << std::endl;

      if ((i % 4) == 0)
        par = {{eps, 0.02f, 12.0f}};
      if ((i % 4) == 1)
        par = {{eps, 0.02f, 9.0f}};
      if ((i % 4) == 2)
        par = {{eps, 0.01f, 9.0f}};
      if ((i % 4) == 3)
        par = {{0.7f * eps, 0.01f, 9.0f}};

      uint32_t nv = 0;
      print<<<1, 1, 0, stream>>>(onGPU_d.view(), ws_d.view());
      cudaCheck(hipGetLastError());
      hipDeviceSynchronize();

      cms::cuda::launch(gpuVertexFinder::clusterTracksByDensityKernel,
                        {1, 512 + 256},
                        onGPU_d.view(),
                        ws_d.view(),
                        kk,
                        par[0],
                        par[1],
                        par[2]);
      print<<<1, 1, 0, stream>>>(onGPU_d.view(), ws_d.view());

      cudaCheck(hipGetLastError());
      hipDeviceSynchronize();

      cms::cuda::launch(gpuVertexFinder::fitVerticesKernel, {1, 1024 - 256}, onGPU_d.view(), ws_d.view(), 50.f);
      cudaCheck(hipGetLastError());
      cudaCheck(hipMemcpy(&nv, &onGPU_d.view().nvFinal(), sizeof(uint32_t), hipMemcpyDeviceToHost));

      if (nv == 0) {
        std::cout << "NO VERTICES???" << std::endl;
        continue;
      }

      float* zv = nullptr;
      float* wv = nullptr;
      float* ptv2 = nullptr;
      int32_t* nn = nullptr;
      uint16_t* ind = nullptr;

      // keep chi2 separated...
      float chi2[2 * nv];  // make space for splitting...

      float hzv[2 * nv];
      float hwv[2 * nv];
      float hptv2[2 * nv];
      int32_t hnn[2 * nv];
      uint16_t hind[2 * nv];

      zv = hzv;
      wv = hwv;
      ptv2 = hptv2;
      nn = hnn;
      ind = hind;

      cudaCheck(hipMemcpy(nn, onGPU_d.view().ndof(), nv * sizeof(int32_t), hipMemcpyDeviceToHost));
      cudaCheck(hipMemcpy(chi2, onGPU_d.view().chi2(), nv * sizeof(float), hipMemcpyDeviceToHost));

      for (auto j = 0U; j < nv; ++j)
        if (nn[j] > 0)
          chi2[j] /= float(nn[j]);
      {
        auto mx = std::minmax_element(chi2, chi2 + nv);
        std::cout << "after fit nv, min max chi2 " << nv << " " << *mx.first << ' ' << *mx.second << std::endl;
      }

      cms::cuda::launch(gpuVertexFinder::fitVerticesKernel, {1, 1024 - 256}, onGPU_d.view(), ws_d.view(), 50.f);
      cudaCheck(hipMemcpy(&nv, &onGPU_d.view().nvFinal(), sizeof(uint32_t), hipMemcpyDeviceToHost));
      cudaCheck(hipMemcpy(nn, onGPU_d.view().ndof(), nv * sizeof(int32_t), hipMemcpyDeviceToHost));
      cudaCheck(hipMemcpy(chi2, onGPU_d.view().chi2(), nv * sizeof(float), hipMemcpyDeviceToHost));

      for (auto j = 0U; j < nv; ++j)
        if (nn[j] > 0)
          chi2[j] /= float(nn[j]);
      {
        auto mx = std::minmax_element(chi2, chi2 + nv);
        std::cout << "before splitting nv, min max chi2 " << nv << " " << *mx.first << ' ' << *mx.second << std::endl;
      }

      // one vertex per block!!!
      cms::cuda::launch(gpuVertexFinder::splitVerticesKernel, {1024, 64}, onGPU_d.view(), ws_d.view(), 9.f);
      cudaCheck(hipMemcpy(&nv, &ws_d.view().nvIntermediate(), sizeof(uint32_t), hipMemcpyDeviceToHost));
      std::cout << "after split " << nv << std::endl;

      cms::cuda::launch(gpuVertexFinder::fitVerticesKernel, {1, 1024 - 256}, onGPU_d.view(), ws_d.view(), 5000.f);
      cudaCheck(hipGetLastError());

      cms::cuda::launch(gpuVertexFinder::sortByPt2Kernel, {1, 256}, onGPU_d.view(), ws_d.view());
      cudaCheck(hipGetLastError());
      cudaCheck(hipMemcpy(&nv, &onGPU_d.view().nvFinal(), sizeof(uint32_t), hipMemcpyDeviceToHost));

      if (nv == 0) {
        std::cout << "NO VERTICES???" << std::endl;
        continue;
      }

      cudaCheck(hipMemcpy(zv, onGPU_d.view().zv(), nv * sizeof(float), hipMemcpyDeviceToHost));
      cudaCheck(hipMemcpy(wv, onGPU_d.view().wv(), nv * sizeof(float), hipMemcpyDeviceToHost));
      cudaCheck(hipMemcpy(chi2, onGPU_d.view().chi2(), nv * sizeof(float), hipMemcpyDeviceToHost));
      cudaCheck(hipMemcpy(ptv2, onGPU_d.view().ptv2(), nv * sizeof(float), hipMemcpyDeviceToHost));
      cudaCheck(hipMemcpy(nn, onGPU_d.view().ndof(), nv * sizeof(int32_t), hipMemcpyDeviceToHost));
      cudaCheck(hipMemcpy(ind, onGPU_d.view().sortInd(), nv * sizeof(uint16_t), hipMemcpyDeviceToHost));
      for (auto j = 0U; j < nv; ++j)
        if (nn[j] > 0)
          chi2[j] /= float(nn[j]);
      {
        auto mx = std::minmax_element(chi2, chi2 + nv);
        std::cout << "nv, min max chi2 " << nv << " " << *mx.first << ' ' << *mx.second << std::endl;
      }

      {
        auto mx = std::minmax_element(wv, wv + nv);
        std::cout << "min max error " << 1. / std::sqrt(*mx.first) << ' ' << 1. / std::sqrt(*mx.second) << std::endl;
      }

      {
        auto mx = std::minmax_element(ptv2, ptv2 + nv);
        std::cout << "min max ptv2 " << *mx.first << ' ' << *mx.second << std::endl;
        std::cout << "min max ptv2 " << ptv2[ind[0]] << ' ' << ptv2[ind[nv - 1]] << " at " << ind[0] << ' '
                  << ind[nv - 1] << std::endl;
      }

      float dd[nv];
      for (auto kv = 0U; kv < nv; ++kv) {
        auto zr = zv[kv];
        auto md = 500.0f;
        for (auto zs : ev.ztrack) {
          auto d = std::abs(zr - zs);
          md = std::min(d, md);
        }
        dd[kv] = md;
      }
      if (i == 6) {
        for (auto d : dd)
          std::cout << d << ' ';
        std::cout << std::endl;
      }
      auto mx = std::minmax_element(dd, dd + nv);
      float rms = 0;
      for (auto d : dd)
        rms += d * d;
      rms = std::sqrt(rms) / (nv - 1);
      std::cout << "min max rms " << *mx.first << ' ' << *mx.second << ' ' << rms << std::endl;

    }  // loop on events
  }  // lopp on ave vert

  return 0;
}
